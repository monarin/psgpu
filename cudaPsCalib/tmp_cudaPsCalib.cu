
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <unistd.h>

#include <sys/time.h>
#include <iostream>
#include <iomanip>
using namespace std;

#define N_PIXELS 2296960
#define SECTOR_SIZE 71780
#define MAX_QUADS 4
#define MAX_SECTORS 8
#define THREADS 256

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void kernel(short *a, int offset, short *dark, int *blockSum)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int i = offset + tid;
  int iDark = i % N_PIXELS;
  a[i] = 1;
  
  // calculate sum per block
  __shared__ int partials[THREADS];
  partials[threadIdx.x] = a[i];
  __syncthreads();

  int j = blockDim.x / 2;
  while (j != 0) {
    if (threadIdx.x < j)
      partials[threadIdx.x] += partials[threadIdx.x + j];
    __syncthreads();
    j /= 2; 
  }
  
  int iBlock = floor( (double) i / blockDim.x );
  blockSum[iBlock] = partials[0];
  //atomicAdd(&blockSum[iBlock], a[i]);
}

__global__ void common_mode(int *blockSum, int offset, int *sectorSum)
{
  int i = offset + threadIdx.x + blockIdx.x * blockDim.x;

  // calculate sector sum
  int iSector = floor( (double) i / blockDim.x );
  atomicAdd(&sectorSum[iSector], blockSum[i]);
}

__global__ void common_mode_apply(short *a, int offset, int *sectorSum)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  int iSector = floor( (double) i / SECTOR_SIZE );
  a[i] = a[i] - (sectorSum[iSector] / SECTOR_SIZE);
}

/* ---------------------- host code -----------------------------*/
void fill( short *p, int n, int val ) {
  for(int i = 0; i < n; i++){
    p[i] = val;
  }
}

float maxError(short *a, int n)
{
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i]-1.0f);
    if (error > maxE) maxE = error;
  }
  return maxE;
}

void host_calc(short *a, short *dark, int *sectorSum, int n) {
  // host calculation
  struct timeval start, end;

  long seconds, useconds;
  double mtime;

  gettimeofday(&start, NULL);
  
  // dark subtraction
  for(int i=0; i<n; i++)
    a[i] -= dark[i];

  // common mode
  for(int i=0; i < MAX_QUADS * MAX_SECTORS; i++) {
    int offset = i * SECTOR_SIZE;
    for(int j=0; j< SECTOR_SIZE; j++) {
      sectorSum[i] += a[offset + j];
    }
  }
  for(int i=0; i < n; i++) {
    int iSector = floor(i / SECTOR_SIZE);
    a[i] -= sectorSum[iSector] / SECTOR_SIZE;
  }

  gettimeofday(&end, NULL);

  seconds  = end.tv_sec  - start.tv_sec;
  useconds = end.tv_usec - start.tv_usec;
  mtime = ((seconds) * 1000000 + useconds)/1000.0;// + 0.5;

  cout << "Host calculation took "<< mtime <<" ms for 1 event."<< endl;
}

int main(int argc, char **argv)
{
  const int maxQuads = 4, maxSectors = 8;
  const int nColumns = 185, nRows = 388;
  const int nPixels = nColumns * nRows * maxSectors * maxQuads;
  const int nEvents = atoi(argv[1]);
  const int n = nPixels * nEvents;

  int nStreams = 32;
  if (argc > 2) nStreams = atoi(argv[2]);
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(short);
  
  const int bytes = n * sizeof(short);
  
  const int darkBytes = nPixels * sizeof(short);
  
  const int blockSize = 256;
  const int nBlocks = n / blockSize;
  const int blockSumBytes = nBlocks * sizeof(int);
  
  const int nSectors = nBlocks / nRows;   
  const int sectorSumBytes = nSectors * sizeof(int);

  printf("Running with nStreams: %d streamSize: %d\n", nStreams, streamSize);
  int gridSize = streamSize / blockSize;
  printf("blockSize: %d gridSize: %d\n", blockSize, gridSize);
  
  int devId = 0;
  if (argc > 3) devId = atoi(argv[3]);
  
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  short *a, *d_a; // data
  checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) ); // host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); // device  
  
  short *dark, *d_dark; // dark
  checkCuda( hipHostMalloc((void**)&dark, darkBytes, hipHostMallocDefault) ); 
  checkCuda( hipMalloc((void**)&d_dark, darkBytes) ); 
  
  int *d_blockSum, *blockSum; // sum of each block
  checkCuda( hipMalloc((void**)&d_blockSum, blockSumBytes) ); 
  checkCuda( hipHostMalloc((void**)&blockSum, blockSumBytes, hipHostMallocDefault) );
  hipMemset(d_blockSum, 0, blockSumBytes);
  
  int *d_sectorSum, *sectorSum; // sum of each sector
  checkCuda( hipMalloc((void**)&d_sectorSum, sectorSumBytes) );
  checkCuda( hipHostMalloc((void**)&sectorSum, sectorSumBytes, hipHostMallocDefault) );
  hipMemset(d_sectorSum, 0, sectorSumBytes);
  
  // prepare raw and dark data
  fill(a, n, 2);
  fill(dark, nPixels, 1);
  memset(sectorSum, 0, sectorSumBytes);
  memset(blockSum, 0, blockSumBytes);

  printf("Input values (Data): %d %d %d...%d %d %d\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  printf("Input values (Dark): %d %d %d...%d %d %d\n", dark[0], dark[1], dark[2], dark[nPixels-3], dark[nPixels-2], dark[nPixels-1]);
  
  // host calculation
  //host_calc(a, dark, sectorSum, nPixels);

  // serial copy for one dark 
  checkCuda( hipMemcpy(d_dark, dark, darkBytes, hipMemcpyHostToDevice) );

  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );

  // asynchronous version 1: loop over {copy, kernel, copy}
  checkCuda( hipEventRecord(startEvent, 0) );
  hipProfilerStart();
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    int offsetSector = i * (streamSize / blockSize);
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
    kernel<<<gridSize, blockSize, 0, stream[i]>>>(d_a, offset, d_dark, d_blockSum);
    //common_mode<<<nBlocks/(nStreams * nRows), nRows, 0, stream[i]>>>(d_blockSum, offsetSector, d_sectorSum); 
    //common_mode_apply<<<gridSize, blockSize, 0, stream[i]>>>(d_a, offset, d_sectorSum);
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  hipProfilerStop(); 
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));
 
  /* 
  cudaMemcpy(blockSum, d_blockSum, blockSumBytes, cudaMemcpyDeviceToHost);
  for (int i = 0; i < nBlocks; i++)
    printf("i=%d blockSum[i]=%d\n", i, blockSum[i]);
  */

  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);
  //cudaFree(d_dark);
  //cudaFreeHost(dark);

  return 0;
}
