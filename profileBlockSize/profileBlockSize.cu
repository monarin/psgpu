
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <unistd.h>

#include <sys/time.h>
#include <iostream>
#include <iomanip>
using namespace std;

#define N_PIXELS 2400000

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void kernel(short *a, int offset, short *dark)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  int iDark = i % N_PIXELS;
  a[i] -= dark[iDark];
}

/* ---------------------- host code -----------------------------*/
void fill( short *p, int n, int val ) {
  for(int i = 0; i < n; i++){
    p[i] = val;
  }
}

float maxError(short *a, int n)
{
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i]-1.0f);
    if (error > maxE) maxE = error;
  }
  return maxE;
}

int main(int argc, char **argv)
{
  const int nPixels = 2400000;				// no. of pixels per image
  const int nEvents = atoi(argv[1]);			// no. of events
  const int n = nPixels * nEvents;			// total number of pixels
  
  const int nStreams = atoi(argv[2]);			// no. of stream
  const int streamSize = n / nStreams;			// stream size (pixels)

  const int streamBytes = streamSize * sizeof(short);	// stream size (bytes)
  const int bytes = n * sizeof(short);			// total size (bytes)
  const int darkBytes = nPixels * sizeof(short);	// dark size (bytes)

  // max block size is 1024
  const int blockSize = atoi(argv[3]);			// block size
  printf("Running with nStreams: %d streamSize: %d\n", nStreams, streamSize);
  int gridSize = streamSize / blockSize;		// grid size
  printf("blockSize: %d gridSize: %d\n", blockSize, gridSize);

  int devId = 0;
  if (argc > 4) devId = atoi(argv[4]);			// device ID (optional)
  
  // print device name
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  short *a, *d_a; 						// data address
  checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) ); 		// host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); 		// device  
  short *dark, *d_dark;					 	// dark address
  checkCuda( hipHostMalloc((void**)&dark, darkBytes, hipHostMallocDefault) ); 	// host pinned
  checkCuda( hipMalloc((void**)&d_dark, darkBytes) );		// device
  
  // prepare data (all 1's) and dark (all 0's) on host
  fill(a, n, 1);
  fill(dark, nPixels, 0);
  printf("Input values (Data): %d %d %d...%d %d %d\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  printf("Input values (Dark): %d %d %d...%d %d %d\n", dark[0], dark[1], dark[2], dark[nPixels-3], dark[nPixels-2], dark[nPixels-1]);

  // host calculation
    struct timeval start, end;

    long seconds, useconds;    
    double mtime;

    gettimeofday(&start, NULL);

    for(int i=0; i<nPixels; i++)
      a[i] -= dark[i];

    gettimeofday(&end, NULL);

    seconds  = end.tv_sec  - start.tv_sec;
    useconds = end.tv_usec - start.tv_usec;
    mtime = ((seconds) * 1000000 + useconds)/1000.0;// + 0.5;

    cout << "Host dark-subtraction took "<< mtime <<" ms for 1 event."<< endl;

  // serial copy for one dark to device 
  checkCuda( hipMemcpy(d_dark, dark, darkBytes, hipMemcpyHostToDevice) );

  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );

  // asynchronous version 1: loop over {copy, kernel, copy}
  checkCuda( hipEventRecord(startEvent, 0) );
  hipProfilerStart();
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
    kernel<<<gridSize, blockSize, 0, stream[i]>>>(d_a, offset, d_dark);
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  hipProfilerStop(); 
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n)); 
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);
  hipFree(d_dark);
  hipHostFree(dark);

  return 0;
}
