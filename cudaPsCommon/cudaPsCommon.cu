
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <unistd.h>

#include <sys/time.h>
#include <iostream>
#include <iomanip>
using namespace std;

#include <string>
#include <sstream>
#include <fstream>

#define N_PIXELS 2296960
#define MAX_QUADS 4
#define MAX_SECTORS 8
#define SECTOR_SIZE 71780

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void kernel(float *a, int offset, float *dark, int streamSize, float *blockSum)
{
  int idx = threadIdx.x + blockIdx.x*blockDim.x; // thread id
  
  // check if idx < streamSize
  if (idx < streamSize) {
 
    int iData = offset + idx;
    int iDark = iData % N_PIXELS;
    a[iData] -= dark[iDark];

    // calculate block sum
    int iBlock = floor( (double) iData / blockDim.x );
    atomicAdd(&blockSum[iBlock], a[iData]);
  }
}

/* ---------------------- host code -----------------------------*/
void fill( float *p, int n, float val ) {
  for(int i = 0; i < n; i++){
    p[i] = val;
  }
}

float maxError(float *aCalc, float *aKnown, int nEvents, int nPixels)
{
  float maxE = 0;
  for (int i = 0; i < nEvents; i++) {
    int offset = i * nPixels;
    for (int j = 0; j < nPixels; j++) {
      int idx = offset + j;
      float error = fabs(aCalc[idx]-aKnown[j]);
      //if (error > 1.0)
      //printf("offset: %d j: %d idx: %d error %e aCalc[idx]: %8.2f aKnown[j]: %8.2f\n", offset, j, idx, error, aCalc[idx], aKnown[j]);
      if (error > maxE) maxE = error;
    }
  }
  return maxE;
}


/* left is the index of the leftmost element of the subarray; right is one
 * past the index of the rightmost element */
void merge_helper(float *input, int left, int right, float *scratch)
{
  /* base case: one element */
  if (right == left + 1) {
    
    return;
  } else {
    
    int i = 0;
    int length = right - left;
    int midpoint_distance = length / 2;
    /* l and r are to the positions in the left and right subarrays */
    int l = left, r = left + midpoint_distance;

    /* sort each subarray */
    merge_helper(input, left, left + midpoint_distance, scratch);
    merge_helper(input, left + midpoint_distance, right, scratch);

    /* merge the arrays together using scratch for temporary storage */
    for (i = 0; i < length; i++) {
      /* Check to see if any elements remain in the left array; if so,
       * we check if there are any elements left in the right array; if
       * so, we compare them. Otherwise, we know that the merge must
       * take the element from the left array */
      if (l < left + midpoint_distance &&
          (r == right || max(input[l], input[r]) == input[l])) {
        
        scratch[i] = input[l];
        l++;
      } else {

        scratch[i] = input[r];
        r++;
      }
    }   
    
    /* Copy the sorted subarray back to the input */
    for(i = left; i < right; i++) {
      input[i] = scratch[i - left];
    }
  }
}

int compare (const void * a, const void * b)
{
  float fa = *(const float*) a;
  float fb = *(const float*) b;
  return (fa > fb) - (fa < fb);
}

void host_calc(float *a, float *dark, int nPixels, int cmmThr) {
  // host calculation
  struct timeval start, end;

  long seconds, useconds;
  double mtime;

  gettimeofday(&start, NULL);
  
  // dark
  for(int i = 0; i < nPixels; i++)
    a[i] -= dark[i];

  // common mode 
  float *sectorMedian = (float *)malloc(MAX_QUADS * MAX_SECTORS * sizeof(float));
  for (int i = 0; i < MAX_QUADS * MAX_SECTORS; i++) {
    
    int offset = i * SECTOR_SIZE;
    
    // select only this sector and sort this sector
    float *sector = (float *)malloc(SECTOR_SIZE * sizeof(float));
    for (int j = 0; j < SECTOR_SIZE; j++) {
      sector[j] = a[offset + j]; 
    }

    //printf("\n");
    //printf("s[0]=%6.2f, s[1]=%6.2f, s[2]=%6.2f\n", sector[0], sector[1], sector[2]);
    /*
    float *scratch = (float *)malloc(SECTOR_SIZE * sizeof(float));
    if (scratch != NULL) {

      merge_helper(sector, 0, SECTOR_SIZE, scratch);
      free(scratch);
      printf("%6.2f, %6.2f, %6.2f ... %6.2f, %6.2f, %6.2f\n", sector[0], sector[1], sector[2], sector[SECTOR_SIZE-3], sector[SECTOR_SIZE-2], sector[SECTOR_SIZE-1]); 
    }*/
    
    qsort(sector, SECTOR_SIZE, sizeof(float), compare);
    //printf("%6.2f, %6.2f, %6.2f ... %6.2f, %6.2f, %6.2f\n", sector[0], sector[1], sector[2], sector[SECTOR_SIZE-3], sector[SECTOR_SIZE-2], sector[SECTOR_SIZE-1]);
    
    // apply the threshold
    int foundPos = 0;
    for (int j = SECTOR_SIZE - 1; j >= 0; j--) {
      if (sector[j] <= cmmThr) {
        foundPos = j;
        break;
      }
      if (j == 0) foundPos = SECTOR_SIZE - 1;
    }   
    
    // calculate median
    if(foundPos%2 == 0) {
      sectorMedian[i] = (sector[foundPos/2] + sector[foundPos/2 - 1]) / 2.0;
    } else {
      sectorMedian[i] = sector[foundPos/2];
    } 
    free(sector);
    printf("sector: %d foundPos: %d med: %6.4f\n", i, foundPos, sectorMedian[i]); 
    
  }

  // apply common mode
  for(int i=0; i < nPixels; i++) {
    int iSector = floor(i / SECTOR_SIZE);
    a[i] -= sectorMedian[iSector];
  }
  
  gettimeofday(&end, NULL);

  seconds  = end.tv_sec  - start.tv_sec;
  useconds = end.tv_usec - start.tv_usec;
  mtime = ((seconds) * 1000000 + useconds)/1000.0;// + 0.5;

  cout << "Host dark-subtraction and common mode took "<< mtime <<" ms for 1 event."<< endl;
}

int main(int argc, char **argv)
{
  const int nPixels = N_PIXELS;			// no. of pixels per image
  const int nEvents = atoi(argv[1]);			// no. of events
  const int n = nPixels * nEvents;			// total number of pixels
  
  const int nStreams = atoi(argv[2]);			// no. of stream
  const int blockSize = atoi(argv[3]);                  // block size (max is 1024)

  const int bytes = n * sizeof(float);			// total size (bytes)
  const int darkBytes = nPixels * sizeof(float);	// dark size (bytes)A

  const int nBlocks = ceil( (double) n / blockSize );
  const int blockSumBytes = nBlocks * sizeof(float);

  const int nSectors = MAX_QUADS * MAX_SECTORS * nEvents;
  const int sectorSumBytes = nSectors * sizeof(float);

  const int cmmThr = 10;
  
  int devId = 0;
  if (argc > 4) devId = atoi(argv[4]);			// device ID (optional)
  
  // print device name
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  // RAW * nEVents
  float *a, *d_a; 						// data address
  checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) ); 		// host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); 		// device
  // SINGLE RAW
  float *raw;                                               // data address
  checkCuda( hipHostMalloc((void**)&raw, darkBytes, hipHostMallocDefault) );     // host pinned
  // RAW-PEDESTAL
  float *pedCorrected, *d_pedCorrected; 			// data address
  checkCuda( hipHostMalloc((void**)&pedCorrected, darkBytes, hipHostMallocDefault) ); // host pinned
  checkCuda( hipMalloc((void**)&d_pedCorrected, darkBytes) ); 	// device  
  // PEDESTAL
  float *dark, *d_dark;					 	// dark address
  checkCuda( hipHostMalloc((void**)&dark, darkBytes, hipHostMallocDefault) ); 	// host pinned
  checkCuda( hipMalloc((void**)&d_dark, darkBytes) );		// device
  // PER-PIXEL GAIN
  float *gain, *d_gain;					 	// dark address
  checkCuda( hipHostMalloc((void**)&gain, darkBytes, hipHostMallocDefault) ); 	// host pinned
  checkCuda( hipMalloc((void**)&d_gain, darkBytes) );		// device
  // RAW-PEDESTAL
  float *calib, *d_calib;					 	// dark address
  checkCuda( hipHostMalloc((void**)&calib, darkBytes, hipHostMallocDefault) ); 	// host pinned
  checkCuda( hipMalloc((void**)&d_calib, darkBytes) );		// device  
  // Sum of each block
  float *d_blockSum, *blockSum; 
  checkCuda( hipMalloc((void**)&d_blockSum, blockSumBytes) );
  checkCuda( hipHostMalloc((void**)&blockSum, blockSumBytes, hipHostMallocDefault) );
  hipMemset(d_blockSum, 0, blockSumBytes);
  // Sum of each sector
  float *d_sectorSum, *sectorSum; 
  checkCuda( hipMalloc((void**)&d_sectorSum, sectorSumBytes) );
  checkCuda( hipHostMalloc((void**)&sectorSum, sectorSumBytes, hipHostMallocDefault) );
  hipMemset(d_sectorSum, 0, sectorSumBytes);

  
  //load the text file and put it into a single string:
  ifstream inR("/reg/data/ana14/cxi/cxitut13/res/yoon82/psgpu/profileBlockSize/cxid9114_raw_95.txt");
  ifstream inPC("/reg/data/ana14/cxi/cxitut13/res/yoon82/psgpu/profileBlockSize/cxid9114_pedCorrected_95.txt");
  ifstream inP("/reg/data/ana14/cxi/cxitut13/res/yoon82/psgpu/profileBlockSize/cxid9114_pedestal_95.txt");
  ifstream inG("/reg/data/ana14/cxi/cxitut13/res/yoon82/psgpu/profileBlockSize/cxid9114_gain_95.txt");
  ifstream inC("/reg/data/ana14/cxi/cxitut13/res/yoon82/psgpu/profileBlockSize/cxid9114_calib_95.txt");
  // Fill arrays from text files
  string line;
  for (unsigned int i=0; i<nPixels; i++){
    getline(inR, line);
    raw[i] = atof(line.c_str());
    //populate all events with the same set of test data
    for (int j=0; j<nEvents; j++) {
      int offset = j * nPixels;
      a[offset + i] = raw[i];
    }
    getline(inPC, line);
    pedCorrected[i] = atof(line.c_str());
    getline(inP, line);
    dark[i] = atof(line.c_str());
    getline(inG, line);
    gain[i] = atof(line.c_str());
    getline(inC, line);
    calib[i] = atof(line.c_str());
  }

  printf("Input values (Data): %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  printf("Input values (Dark): %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", dark[0], dark[1], dark[2], dark[nPixels-3], dark[nPixels-2], dark[nPixels-1]);

  // host calculation 
  host_calc(raw, dark, nPixels, cmmThr);

  /*
  printf("Host Calculation\n");
  printf("Input values (Data calc.): %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", raw[0], raw[1], raw[2], raw[nPixels-3], raw[nPixels-2], raw[nPixels-1]);
  printf("Input values (Data known): %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", calib[0], calib[1], calib[2], calib[nPixels-3], calib[nPixels-2], calib[nPixels-1]);
  printf("  max error: %e\n", maxError(raw, calib, 1, nPixels));

  // serial copy for one dark to device 
  checkCuda( cudaMemcpy(d_dark, dark, darkBytes, cudaMemcpyHostToDevice) );

  float ms; // elapsed time in milliseconds

  // create events and streams
  cudaEvent_t startEvent, stopEvent, dummyEvent;
  cudaStream_t stream[nStreams];
  checkCuda( cudaEventCreate(&startEvent) );
  checkCuda( cudaEventCreate(&stopEvent) );
  checkCuda( cudaEventCreate(&dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( cudaStreamCreate(&stream[i]) );

  // asynchronous version 1: loop over {copy, kernel, copy}
  checkCuda( cudaEventRecord(startEvent, 0) );
  cudaProfilerStart();
  for (int i = 0; i < nStreams; ++i) {
    int streamSize = ceil( (double) n / nStreams );  // stream size (pixels)
    int offset = i * streamSize;

    // check if last stream has full length
    if ( (i + 1) * streamSize > n ) streamSize = n - (i * streamSize);

    int streamBytes = streamSize * sizeof(float);   // stream size (bytes)
    //printf("Stream#: %d streamSize: %d offset=%d\n", i, streamSize, offset);
    int gridSize = ceil(  (double) streamSize / blockSize );               // grid size
    //printf("blockSize: %d gridSize: %d\n", blockSize, gridSize);

    checkCuda( cudaMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, cudaMemcpyHostToDevice,
                               stream[i]) );
    kernel<<<gridSize, blockSize, 0, stream[i]>>>(d_a, offset, d_dark, streamSize, d_blockSum);
    checkCuda( cudaMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, cudaMemcpyDeviceToHost,
                               stream[i]) );
  }
  cudaProfilerStop(); 
  checkCuda( cudaEventRecord(stopEvent, 0) );
  checkCuda( cudaEventSynchronize(stopEvent) );
  checkCuda( cudaEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("GPU Calculation\n");
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("Input values (Data calc.): %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  printf("Input values (Data known): %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", pedCorrected[0], pedCorrected[1], pedCorrected[2], pedCorrected[nPixels-3], pedCorrected[nPixels-2], pedCorrected[nPixels-1]);
  printf("  max error: %e\n", maxError(a, pedCorrected, nEvents, nPixels));
     
  //cudaMemcpy(blockSum, d_blockSum, blockSumBytes, cudaMemcpyDeviceToHost);
  //for (int i = 0; i < nBlocks; i++)
  //  printf("i=%d blockSum[i]=%10.2f\n", i, blockSum[i]);
  //
  // cleanup
  checkCuda( cudaEventDestroy(startEvent) );
  checkCuda( cudaEventDestroy(stopEvent) );
  checkCuda( cudaEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( cudaStreamDestroy(stream[i]) );
  cudaFree(d_a);
  cudaFreeHost(a);
  cudaFree(d_dark);
  cudaFreeHost(dark);
  */
  return 0;
}
