#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <unistd.h>

#include <sys/time.h>
#include <iostream>
#include <iomanip>
using namespace std;

#include <string>
#include <sstream>
#include <fstream>

#include "cudaPsCommon.h"

#define N_PIXELS 2296960
#define MAX_QUADS 4
#define MAX_SECTORS 8
#define SECTOR_SIZE 71780

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__global__ void kernel(float *a, 
                       int offset, 
                       float *dark, 
                       short *bad, 
                       float cmmThr,
                       int streamSize, 
                       float *blockSum,
                       int *cnBlockSum)
{
  int idx = threadIdx.x + blockIdx.x*blockDim.x; // thread id
  
  // check if idx < streamSize
  if (idx < streamSize) {
 
    int iData = offset + idx;
    int iDark = iData % N_PIXELS;
    a[iData] -= dark[iDark];

    // calculate block sum
    if ( bad[iDark] == 0 && a[iData] < cmmThr) {
      int iBlock = floor( (double) iData / blockDim.x );
      atomicAdd(&blockSum[iBlock], a[iData]);
      atomicAdd(&cnBlockSum[iBlock], 1);
    }
  }
}

__global__ void common_mode(float *blockSum, int *cnBlockSum, float *sectorSum, int *cnSectorSum, int offset)
{
  int i = offset + threadIdx.x + blockIdx.x * blockDim.x;

  // calculate sector sum and sector count
  int iSector = floor( (double) i / blockDim.x );
  atomicAdd(&sectorSum[iSector], blockSum[i]);
  atomicAdd(&cnSectorSum[iSector], cnBlockSum[i]);
}

__global__ void common_mode_apply(float *a, float *sectorSum, int *cnSectorSum, float *gain, int offset)
{
  int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
  int iGain = i % N_PIXELS;
  int iSector = floor( (double) i / SECTOR_SIZE );
  a[i] = ( a[i] - (sectorSum[iSector] / cnSectorSum[iSector]) ) * gain[iGain];
}
   

/* ---------------------- host code -----------------------------*/
void fill( float *p, int n, float val ) {
  for(int i = 0; i < n; i++){
    p[i] = val;
  }
}

float maxError(float *aCalc, float *aKnown, int nEvents, int nPixels)
{
  float maxE = 0;
  for (int i = 0; i < nEvents; i++) {
    int offset = i * nPixels;
    for (int j = 0; j < nPixels; j++) {
      int idx = offset + j;
      float error = fabs(aCalc[idx]-aKnown[j]);
      //if (error > 1.0)
      //printf("offset: %d j: %d idx: %d error %e aCalc[idx]: %8.2f aKnown[j]: %8.2f\n", offset, j, idx, error, aCalc[idx], aKnown[j]);
      if (error > maxE) maxE = error;
    }
  }
  return maxE;
}

// used in host_calculation qsort function
int compare (const void * a, const void * b)
{
  float fa = *(const float*) a;
  float fb = *(const float*) b;
  return (fa > fb) - (fa < fb);
}

// host-side calculation comparision
void host_calc(float *a, float *dark, int nPixels, int cmmThr) {
  // host calculation
  struct timeval start, end;

  long seconds, useconds;
  double mtime;

  gettimeofday(&start, NULL);
  
  // dark
  for(int i = 0; i < nPixels; i++)
    a[i] -= dark[i];

  // common mode 
  float *sectorMedian = (float *)malloc(MAX_QUADS * MAX_SECTORS * sizeof(float));
  for (int i = 0; i < MAX_QUADS * MAX_SECTORS; i++) {
    
    int offset = i * SECTOR_SIZE;
    
    // select only this sector and sort this sector
    float *sector = (float *)malloc(SECTOR_SIZE * sizeof(float));
    for (int j = 0; j < SECTOR_SIZE; j++) {
      sector[j] = a[offset + j]; 
    }

    //printf("\n");
    //printf("s[0]=%6.2f, s[1]=%6.2f, s[2]=%6.2f\n", sector[0], sector[1], sector[2]);
    
    qsort(sector, SECTOR_SIZE, sizeof(float), compare);
    //printf("%6.2f, %6.2f, %6.2f ... %6.2f, %6.2f, %6.2f\n", sector[0], sector[1], sector[2], sector[SECTOR_SIZE-3], sector[SECTOR_SIZE-2], sector[SECTOR_SIZE-1]);
    
    // apply the threshold
    int foundPos = 0;
    for (int j = SECTOR_SIZE - 1; j >= 0; j--) {
      if (sector[j] <= cmmThr) {
        foundPos = j;
        break;
      }
      if (j == 0) foundPos = SECTOR_SIZE - 1;
    }   
    
    // calculate median
    if(foundPos%2 == 0) {
      sectorMedian[i] = (sector[foundPos/2] + sector[foundPos/2 - 1]) / 2.0;
    } else {
      sectorMedian[i] = sector[foundPos/2];
    } 
    free(sector);
    printf("sector: %d foundPos: %d med: %6.4f \n", i, foundPos, sectorMedian[i]); 
    
  }

  // apply common mode
  for(int i=0; i < nPixels; i++) {
    int iSector = floor(i / SECTOR_SIZE);
    a[i] -= sectorMedian[iSector];
  }
  
  gettimeofday(&end, NULL);

  seconds  = end.tv_sec  - start.tv_sec;
  useconds = end.tv_usec - start.tv_usec;
  mtime = ((seconds) * 1000000 + useconds)/1000.0;// + 0.5;

  cout << "Host dark-subtraction and common mode took "<< mtime <<" ms for 1 event."<< endl;
}

int main(int argc, char **argv)
{
  const int nPixels = N_PIXELS;			              // no. of pixels per image
  const int nRows = 388;                          // no. of rows in a sector
  const int nEvents = atoi(argv[1]);			        // no. of events
  const int n = nPixels * nEvents;			          // total number of pixels
  
  const int nStreams = atoi(argv[2]);			        // no. of stream
  const int blockSize = atoi(argv[3]);            // block size (max is 1024)

  const int bytes = n * sizeof(float);			      // total size (bytes)
  const int darkBytes = nPixels * sizeof(float);	// dark size (bytes)A

  const int nBlocks = ceil( (double) n / blockSize );
  const int blockSumBytes = nBlocks * sizeof(float);

  const int nSectors = MAX_QUADS * MAX_SECTORS * nEvents;
  const int sectorSumBytes = nSectors * sizeof(float);

  const float cmmThr = 100.0f;
  
  int devId = 0;
  if (argc > 4) devId = atoi(argv[4]);			// device ID (optional)
  
  // print device name
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  // RAW * nEVents
  float *a, *d_a; 						
  checkCuda( hipHostMalloc((void**)&a, bytes) ); 		// host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); 		// device
  // SINGLE RAW
  float *raw;                                               
  checkCuda( hipHostMalloc((void**)&raw, darkBytes) );     // host pinned
  // RAW-PEDESTAL
  float *pedCorrected, *d_pedCorrected; 			
  checkCuda( hipHostMalloc((void**)&pedCorrected, darkBytes) ); // host pinned
  checkCuda( hipMalloc((void**)&d_pedCorrected, darkBytes) ); 	// device  
  // PEDESTAL
  float *dark, *d_dark;					 
  checkCuda( hipHostMalloc((void**)&dark, darkBytes) ); 	// host pinned
  checkCuda( hipMalloc((void**)&d_dark, darkBytes) );		// device
  // PER-PIXEL GAIN
  float *gain, *d_gain;					 	
  checkCuda( hipHostMalloc((void**)&gain, darkBytes) ); 	// host pinned
  checkCuda( hipMalloc((void**)&d_gain, darkBytes) );		// device
  // BAD PIXEL FLAGS
  short *bad, *d_bad;           
  checkCuda( hipHostMalloc((void**)&bad, nPixels * sizeof(short)) );  // host pinned
  checkCuda( hipMalloc((void**)&d_bad, nPixels * sizeof(short)) );    // device
  // RAW-PEDESTAL
  float *calib, *d_calib;					 	
  checkCuda( hipHostMalloc((void**)&calib, darkBytes) ); 	// host pinned
  checkCuda( hipMalloc((void**)&d_calib, darkBytes) );		// device  
  // Sum of each block
  float *d_blockSum, *blockSum; 
  checkCuda( hipMalloc((void**)&d_blockSum, blockSumBytes) );
  checkCuda( hipHostMalloc((void**)&blockSum, blockSumBytes) );
  hipMemset(d_blockSum, 0, blockSumBytes);
  int *d_cnBlockSum;
  checkCuda( hipMalloc((void**)&d_cnBlockSum, nBlocks * sizeof(int)) );
  // Sum of each sector
  float *d_sectorSum, *sectorSum; 
  checkCuda( hipMalloc((void**)&d_sectorSum, sectorSumBytes) );
  checkCuda( hipHostMalloc((void**)&sectorSum, sectorSumBytes) );
  hipMemset(d_sectorSum, 0, sectorSumBytes);
  int * d_cnSectorSum;
  checkCuda( hipMalloc((void**)&d_cnSectorSum, nSectors * sizeof(int)) );
  // Peak centroids
  const int nCenters = FILTER_PATCH_PER_SECTOR * (FILTER_PATCH_WIDTH / FILTER_PATCH_HEIGHT)
                         * nSectors;
  uint *d_centers, *centers;
  checkCuda( hipMalloc((void**)&d_centers, nCenters * sizeof(uint)) );
  checkCuda( hipHostMalloc((void**)&centers, nCenters * sizeof(uint)) );
  hipMemset(d_centers, 0, nCenters * sizeof(uint));
  // Peaks
  int nPeaks = 640;
  Peak *d_peaks = NULL;
  checkCuda( (hipMalloc((void**)&d_peaks, nPeaks * sizeof(Peak))) );
  hipMemset(d_peaks, 0, nPeaks * sizeof(Peak));
  uint *d_conmap;
  checkCuda( (hipMalloc((void**)&d_conmap, n * sizeof(uint))) );
  hipMemset(d_conmap, 0, n * sizeof(uint));

  //load the text file and put it into a single string:
  ifstream inR("/reg/neh/home/monarin/psgpu/data/cxid9114_raw_95_hit01.txt");
  ifstream inPC("/reg/neh/home/monarin/psgpu/data/cxid9114_pedCorrected_95.txt");
  ifstream inP("/reg/neh/home/monarin/psgpu/data/cxid9114_pedestal_95.txt");
  ifstream inG("/reg/neh/home/monarin/psgpu/data/cxid9114_gain_95.txt");
  ifstream inB("/reg/neh/home/monarin/psgpu/data/cxid9114_badpix_fake_95.txt");
  ifstream inC("/reg/neh/home/monarin/psgpu/data/cxid9114_calib_95.txt");
  // Fill arrays from text files
  string line;
  for (unsigned int i=0; i<nPixels; i++){
    getline(inR, line);
    raw[i] = atof(line.c_str());
    //populate all events with the same set of test data
    for (int j=0; j<nEvents; j++) {
      int offset = j * nPixels;
      a[offset + i] = raw[i];
    }
    getline(inPC, line);
    pedCorrected[i] = atof(line.c_str());
    getline(inP, line);
    dark[i] = atof(line.c_str());
    getline(inG, line);
    gain[i] = atof(line.c_str());
    getline(inB, line);
    bad[i] = atoi(line.c_str());
    getline(inC, line);
    calib[i] = atof(line.c_str());
  }
  puts("Input\n");
  printf("Data       : %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  printf("Dark       : %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", dark[0], dark[1], dark[2], dark[nPixels-3], dark[nPixels-2], dark[nPixels-1]);
  printf("Bad pixels : %d %d %d...%d %d %d\n", bad[0], bad[1], bad[2], bad[nPixels-3], bad[nPixels-2], bad[nPixels-1]);
  printf("Pixel gain : %8.2f %8.2f %8.2f ... %8.2f %8.2f %8.2f\n", gain[0], gain[1], gain[2], gain[nPixels-3], gain[nPixels-2], gain[nPixels-1]);
  

  // host calculation 
  /*host_calc(raw, dark, nPixels, cmmThr);

  
  printf("Host Calculation\n");
  printf("Input values (Data calc.): %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", raw[0], raw[1], raw[2], raw[nPixels-3], raw[nPixels-2], raw[nPixels-1]);
  printf("Input values (Data known): %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", calib[0], calib[1], calib[2], calib[nPixels-3], calib[nPixels-2], calib[nPixels-1]);
  printf("  max error: %e\n", maxError(raw, calib, 1, nPixels));
  */
  // 
  // serial copy for one dark, bad pixel mask, and pixel gain to device 
  checkCuda( hipMemcpy(d_dark, dark, darkBytes, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(d_bad, bad, nPixels * sizeof(short), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(d_gain, gain, darkBytes, hipMemcpyHostToDevice) );

  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );

  // asynchronous version 1: loop over {copy, kernel, copy}
  checkCuda( hipEventRecord(startEvent, 0) );
  hipProfilerStart();
  for (int i = 0; i < nStreams; ++i) {
    int streamSize = ceil( (double) n / nStreams );  // stream size (pixels)
    int offset = i * streamSize;
    int offsetSector = i * (streamSize / blockSize);
    int filterPatchStreamSize = FILTER_PATCH_PER_SECTOR * nEvents;
    int offsetFilterPatch = i * filterPatchStreamSize;
    int peakStreamSize = nPeaks / nStreams;
    int offsetPeakStreamSize = i * peakStreamSize;

    // check if last stream has full length
    if ( (i + 1) * streamSize > n ) streamSize = n - (i * streamSize);

    int streamBytes = streamSize * sizeof(float);   // stream size (bytes)
    //printf("Stream#: %d streamSize: %d offset=%d\n", i, streamSize, offset);
    int gridSize = ceil(  (double) streamSize / blockSize );               // grid size
    //printf("blockSize: %d gridSize: %d\n", blockSize, gridSize);

    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );

    // calibration kernels
    kernel<<<gridSize, blockSize, 0, stream[i]>>>(d_a, offset, d_dark, d_bad, cmmThr, streamSize, d_blockSum, d_cnBlockSum);
    common_mode<<<nBlocks/(nStreams * nRows), nRows, 0, stream[i]>>>(d_blockSum, d_cnBlockSum, d_sectorSum, d_cnSectorSum, offsetSector);
    common_mode_apply<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, d_sectorSum, d_cnSectorSum, d_gain, offset); 

    // peakFinder kernels
    filterByThrHigh_v2<<<FILTER_PATCH_PER_SECTOR * nSectors / nStreams, FILTER_THREADS_PER_PATCH, 0, stream[i]>>>(d_a, d_centers, offsetFilterPatch);
    floodFill_v2<<<nPeaks/nStreams, 64, 0, stream[i]>>>(d_a, d_centers, d_peaks, d_conmap, offsetPeakStreamSize, nEvents);

    // copy data out
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  hipProfilerStop(); 
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("GPU Calculation\n");
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("Output               : %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  printf("Known ped. corrected : %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", pedCorrected[0], pedCorrected[1], pedCorrected[2], pedCorrected[nPixels-3], pedCorrected[nPixels-2], pedCorrected[nPixels-1]);
  printf("Know calibrated      : %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", calib[0], calib[1], calib[2],calib[nPixels-3], calib[nPixels-2], calib[nPixels-1]);
  printf("Differences          : %8.2f %8.2f %8.2f...%8.2f %8.2f %8.2f\n", a[0]-calib[0], a[1]-calib[1], a[2]-calib[2], a[n-3]-calib[nPixels-3], a[n-2]-calib[nPixels-2], a[n-1]-calib[nPixels-1]);
  printf("  max error: %e\n", maxError(a, pedCorrected, nEvents, nPixels));
     
  /*hipMemcpy(centers, d_centers, nCenters * sizeof(uint), hipMemcpyDeviceToHost);
  for (int i = 0; i < 500; i++) {
    if (centers[i] > 0) 
      printf("i=%d centers[i]=%d\n", i, centers[i]);
  }*/
  //
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);
  hipFree(d_dark);
  hipHostFree(dark);
  
  return 0;
}
